#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

#define TILE_WIDTH 32

#if 0
#define DEBUG(x...) printf(x)
#else
#define DEBUG(x...) ;
#endif


// see if i, j is a valid index of a matrix of dim m x n.
#define IS_VALID(m, n, i, j) ((i) < (m) && (j) < (n))

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
    // block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // row and column we are working on within the matrix
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;

    int n_phases = (M.width + TILE_WIDTH - 1) / TILE_WIDTH;

    // now, for each phase we need to do two things:
    // 1. move relevant data into shared memory
    // 2. compute partial results

    // allocate shared memory
    // each block gets TILE_WIDTH*TILE_WIDTH threads
    __shared__ float M_entries[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_entries[TILE_WIDTH][TILE_WIDTH];

    // running sum of partial dot products
    float output = 0;

    for (int phase = 0; phase < n_phases; phase++) {
        DEBUG("phase %d\n", phase);
        // move data into shared memory
        int M_col_offset = phase*TILE_WIDTH+tx;
        if (IS_VALID(M.height, M.width, row, M_col_offset)) {
            DEBUG("valid: %d, %d\n", row, M_col_offset);
            M_entries[ty][tx] = M.elements[row*M.width + M_col_offset];
        } else {
            M_entries[ty][tx] = 0;
        }
        int N_row_offset = phase*TILE_WIDTH+ty;
        if (IS_VALID(N.height, N.width, N_row_offset, col)) {
            DEBUG("valid: %d, %d\n", N_row_offset, col);
            N_entries[ty][tx] = N.elements[N_row_offset*N.width + col];
        } else {
            N_entries[ty][tx] = 0;
        }
        // wait for all threads to move data into shared memory before continuing
        __syncthreads();

        // calculate partial dot product/result, standard algorithm...
        for (int k = 0; k < TILE_WIDTH; k++) {
            output += M_entries[ty][k] * N_entries[k][tx];
        }
        __syncthreads();
    }
    
    // only set output if we are accessing valid memory
    if (IS_VALID(M.height, N.width, row, col))
        P.elements[row*N.width+col] = output;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
